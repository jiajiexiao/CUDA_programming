#include "hip/hip_runtime.h"
/*
CSC691 GPU programming
Project 2: In the Interest of Time
Tiled GPU version
Jiajie Xiao
Oct 8, 2017
*/

#include <stdio.h> 
#include <stdlib.h>
#include <time.h>
#define TILE_WIDTH 64
#define BLOCK_DIM 64

void gen_flattened_array(float *array, int rowSize, int colSize)
{
    int i, j;
	time_t t;
	/* Intializes random number generator */
   	srand((unsigned) time(&t));
   	for (i=0;i<rowSize;i++)
   	{
   		for (j=0;j<colSize;j++)
   		{
   			array[i*colSize+j]=1.0*rand()/RAND_MAX;
   		}
   	}
}

void output_array(float *array, int rowSize, int colSize)
{
    int i, j;
    printf("%s\t%s\t%s\n", "RowID", "ColID", "Value");
    for (i=0; i<rowSize; i++)
    {
    	for (j=0;j<colSize;j++)
    	{
    		 printf("%d\t%d\t%f\n", i, j, array[i*colSize+j]);
    	}
    }
}

void save_array(float *array, int rowSize, int colSize, FILE *fp)
{
    int i, j;
    for (i=0; i<rowSize; i++)
    {
        for (j=0;j<colSize;j++)
        {
            fprintf(fp, "%d\t%d\t%f\n", i, j, array[i*colSize+j]);
        }
    }
}

__global__ void compute_matixMult(float *a, float *b, float *c, int rowSize_a, int colSize_a, int colSize_b) // colSize_a should be equal to rowSize_b
{
    __shared__ float ds_a[TILE_WIDTH][TILE_WIDTH]; // sub-a tile
    __shared__ float ds_b[TILE_WIDTH][TILE_WIDTH]; // sub-b tile

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;

    float sum = 0.0;


    // loop over a and b tiles required to compute the element p
    int p;
    for (p = 0; p < (colSize_a - 1)/TILE_WIDTH + 1; p++)
    {
        // Collaborative loading of M and N tiles into shared memory
        if (row < rowSize_a && p*TILE_WIDTH+tx < colSize_a) 
            ds_a[ty][tx] = a[row*colSize_a + p*TILE_WIDTH + tx];
        else
            ds_a[ty][tx] = 0.0;
    
        if (p*TILE_WIDTH + ty < colSize_a && col < colSize_b)
            ds_b[ty][tx] = b[(p*TILE_WIDTH+ty)*colSize_b + col];
        else
            ds_b[ty][tx] = 0.0;
        __syncthreads();

        if (row < rowSize_a && col < colSize_b)
        {
            int i;
            for (i = 0; i < TILE_WIDTH; ++i)
                sum += ds_a[ty][i] * ds_b[i][tx];
        }
        __syncthreads();
    }
    if (row < rowSize_a && col < colSize_b)
        c[row*colSize_b+col] = sum;
}

int main(int argc, char **argv)
{
    int rowSize_A, colSize_A, rowSize_B, colSize_B, rowSize_C, colSize_C;

    if (argc < 2)
    {
        printf("At least one varaible for matrix dimensions is required.");
        return -1;
    }
    else if (argc == 2)
    {	
    	rowSize_A = atoi(argv[1]);
    	colSize_A = atoi(argv[1]);
    	rowSize_B = atoi(argv[1]);
    	colSize_B = atoi(argv[1]);
    	printf("Two random %dX%d matrices are going to be generated for the multiplication.\n", rowSize_A, colSize_A);
    }
    else
    {    	
    	rowSize_A = atoi(argv[1]);
    	colSize_A = atoi(argv[2]);
    	rowSize_B = atoi(argv[3]);
    	colSize_B = atoi(argv[4]);

    	if (colSize_A == rowSize_B)
    		printf("A random %dX%d matrix A and %dX%d matrix B are going to be generated for the multiplication.\n", rowSize_A, colSize_A, rowSize_B, colSize_B);
    	else
    	{
    		printf("Error input: the number of columns of the first matrix should be equal to the number of rows of te second matrix");
    		return -1;
    	}
    }

    /* Generated two random matrices for multiplication */
    rowSize_C = rowSize_A; 
    colSize_C = colSize_B;
    float *matrix_A, *matrix_B, *matrix_C;  
    matrix_A = (float *) malloc(rowSize_A*colSize_A*sizeof(float));
    matrix_B = (float *) malloc(rowSize_B*colSize_B*sizeof(float));
    matrix_C = (float *) malloc(rowSize_C*colSize_C*sizeof(float)); 

    gen_flattened_array(matrix_A, rowSize_A, colSize_A);
    gen_flattened_array(matrix_B, rowSize_B, colSize_B);

    /* Malloc memory for device variables */
    float *dev_matrix_A, *dev_matrix_B, *dev_matrix_C; 
    hipMalloc((void **) & dev_matrix_A, rowSize_A*colSize_A*sizeof(float));
    hipMalloc((void **) & dev_matrix_B, rowSize_B*colSize_B*sizeof(float));
    hipMalloc((void **) & dev_matrix_C, rowSize_C*colSize_C*sizeof(float));

    hipMemcpy(dev_matrix_A, matrix_A, rowSize_A*colSize_A*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrix_B, matrix_B, rowSize_B*colSize_B*sizeof(float), hipMemcpyHostToDevice);

    /* Computate the multiplication */
     dim3 dimBlock (BLOCK_DIM, BLOCK_DIM,1);
     dim3 dimGrid((int)ceil((colSize_B-1)/TILE_WIDTH+1),(int)ceil((rowSize_A-1)/TILE_WIDTH+1),1);

    compute_matixMult<<<dimGrid, dimBlock>>>(dev_matrix_A, dev_matrix_B, dev_matrix_C, rowSize_A, colSize_A, colSize_B);
    hipDeviceSynchronize();

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    compute_matixMult<<<dimGrid, dimBlock>>>(dev_matrix_A, dev_matrix_B, dev_matrix_C, rowSize_A, colSize_A, colSize_B);
    hipDeviceSynchronize();
    hipEventRecord(stop);

    /* transfer data from Device to Host and free device memory */
    hipMemcpy(matrix_C, dev_matrix_C, rowSize_C*colSize_C*sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop);

    hipFree(dev_matrix_A);
    hipFree(dev_matrix_B);
    hipFree(dev_matrix_C);

    hipEventSynchronize(stop);
    float time, milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    time = milliseconds/1000000.0;
    printf("The product is computed.\n Execution time is %.10e seconds\n", time);

    /* Output */
    //printf("Matrix A\n");
    //output_array(matrix_A, rowSize_A, colSize_A);
    //printf("Matrix B\n");
    //output_array(matrix_B, rowSize_B, colSize_B);
    //printf("Product Matrix C\n");    
    //output_array(matrix_C, rowSize_A, colSize_B);

    FILE *file_product, *file_time;
    file_product = fopen("product.txt", "w");
    if (file_product == NULL)
        printf("Error open file to save the product matrix");
    else
    {
        save_array(matrix_C, rowSize_A, colSize_B, file_product);
        fclose(file_product);
    }


    file_time = fopen("time.txt", "a");
    if (file_time == NULL)
        printf("Error open file to save the product matrix");
    else
    {
        fprintf(file_time, "%d\t%.10e\n", colSize_A, time);
        fclose(file_time);
    }

    return 1;

}

