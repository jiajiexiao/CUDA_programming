#include "hip/hip_runtime.h"
/*
CSC691 GPU programming
Project 3: Pi Time
Jiajie Xiao
Oct 23, 2017
*/

#include<stdio.h>
#include <hiprand.h> // cuRAND host and device header files
#include <hiprand/hiprand_kernel.h> 

//#define numsamples 1000000
//#define numThreadPerBlock 1024


// kernel to initialize the random states
__global__ void random_init (unsigned int seed, hiprandState_t *states)
{
	hiprand_init(seed, threadIdx.x, 0, &states[blockIdx.x * blockDim.x + threadIdx.x]);
}


// Generate a random point  
__global__ void random_sample(hiprandState_t * states, unsigned int * SuccessLanded, unsigned int numLoops)
{
	float x, y;
	unsigned int n, idx;
	idx = blockIdx.x * blockDim.x + threadIdx.x;
	SuccessLanded[idx] = 0;

	for (n=0; n<numLoops; n++)
	{
		x = hiprand_uniform(&states[idx]);
		y = hiprand_uniform(&states[idx]);
		//printf("%f\t%f\n", x,y);
		if (x*x+y*y<1) 
			SuccessLanded[idx]++;	
	}
}


int main(int argc, char **argv)
{
	int numsamples;
	if (argc < 2)
    {
    	printf("Please type in number of iterations to compute Pi.\n");
        return -1;
    }
    else
    {
    	numsamples = atoi(argv[1]);
    	if(numsamples<1)
    	{
    		printf("Please type in positive itergers.\n");
    		return -1;
    	}
    }
    int numThreadPerBlock = numsamples>1000?1000:numsamples;


	unsigned int numBlocks, numLoops;
	numBlocks = numsamples/numThreadPerBlock;
	if(numBlocks>100)
		numBlocks =100;
	numLoops = numsamples/(numBlocks*numThreadPerBlock);

	unsigned int *SuccessLanded, *dev_SuccessLanded;
	SuccessLanded = (unsigned int *) malloc(numThreadPerBlock*numBlocks * sizeof(unsigned int));
	hipMalloc((void**)&dev_SuccessLanded, numThreadPerBlock*numBlocks * sizeof(unsigned int));
	hipMemcpy(dev_SuccessLanded, SuccessLanded, numThreadPerBlock*numBlocks * sizeof(unsigned int), hipMemcpyHostToDevice);

	hiprandState_t *states;
	hipMalloc((void**) &states, numThreadPerBlock*numBlocks * sizeof(hiprandState_t));


	// initialize all of the random states on the GPU
	random_init<<<numBlocks,numThreadPerBlock>>>(time(NULL),states);

	// Perform Monte Carlo sampling
	random_sample<<<numBlocks,numThreadPerBlock>>> (states, dev_SuccessLanded, numLoops);

	hipEvent_t start_gpu, stop_gpu;
	hipEventCreate(&start_gpu);
	hipEventCreate(&stop_gpu);

	hipEventRecord(start_gpu);
	// initialize all of the random states on the GPU
	random_init<<<numBlocks,numThreadPerBlock>>>(time(NULL),states);

	// Perform Monte Carlo sampling
	random_sample<<<numBlocks,numThreadPerBlock>>> (states, dev_SuccessLanded, numLoops);
	//hipDeviceSynchronize();
	hipEventRecord(stop_gpu);
	
	hipMemcpy(SuccessLanded, dev_SuccessLanded, numThreadPerBlock*numBlocks * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipFree(dev_SuccessLanded);

	hipEventSynchronize(stop_gpu);
	float milliseconds_gpu = 0;
    hipEventElapsedTime(&milliseconds_gpu, start_gpu, stop_gpu);

	clock_t start_cpu, stop_cpu;
    start_cpu = clock();
	unsigned int numPointsLanded = 0;
	for (unsigned int n=0; n<(numThreadPerBlock*numBlocks); n++)
	{
		numPointsLanded += SuccessLanded[n];	
	}
	stop_cpu = clock();
    float milliseconds_cpu = 1000.0*(stop_cpu-start_cpu)/(float)CLOCKS_PER_SEC;


	unsigned int actualSample = numLoops* numBlocks * numThreadPerBlock;
	printf("%d\t%d\t%d\t%d.\n", numPointsLanded, actualSample, numBlocks, numLoops);
	float computedPi = 4.0*numPointsLanded/(actualSample);
	printf("The computed Pi has a value of %.6f.\n", computedPi);

	FILE *file;
    file = fopen("results.txt", "a");
    if (file == NULL)
        printf("Error open file to save the result file");
    else
    {
        fprintf(file, "%d\t%e\t%e\t%e\n", actualSample, computedPi, milliseconds_gpu, milliseconds_cpu);
        fclose(file);
    }
	
	return 0;
}
